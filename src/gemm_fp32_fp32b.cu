#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "kernel.h"

namespace ape {
void gemm_fp32_fp32b(ApeTrans transa, ApeTrans transb, int m, int n, int k, const float *alpha, const float *A, int lda,
                     const float *B, int ldb, const float *beta, float *C, int ldc) {
    __hip_bfloat16 *buf, *buf_a, *buf_b;
    cudaSafeCall(hipMalloc((void **)&buf, sizeof(__hip_bfloat16) * (m * k + k * n) * 3));
    buf_a = buf;
    buf_b = buf + m * k * 3;

    split_fp32_to_bf16(buf_a, A, m * k);
    split_fp32_to_bf16(buf_b, B, k * n);

    float beta0 = *beta, beta1 = 1;
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, buf_a,
                                HIP_R_16BF, lda, buf_b, HIP_R_16BF, ldb, &beta0, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F,
                                HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha,
                                buf_a + m * k, HIP_R_16BF, lda, buf_b, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc,
                                HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, buf_a,
                                HIP_R_16BF, lda, buf_b + k * n, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc,
                                HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha,
                                buf_a + m * k, HIP_R_16BF, lda, buf_b + k * n, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc,
                                HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha,
                                buf_a + m * k * 2, HIP_R_16BF, lda, buf_b, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc,
                                HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, buf_a,
                                HIP_R_16BF, lda, buf_b + k * n * 2, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc,
                                HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));

    cudaSafeCall(hipFree(buf));
}
} // namespace ape
