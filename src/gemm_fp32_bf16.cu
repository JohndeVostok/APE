#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "kernel.h"

namespace ape
{
void gemm_fp32_bf16(ApeTrans transa, ApeTrans transb, int m, int n, int k, const float *alpha, const float *A, int lda,
                      const float *B, int ldb, const float *beta, float *C, int ldc) {
    __hip_bfloat16 *bf16_A = (__hip_bfloat16*)ape_buffer, *bf16_B = bf16_A + m * k * 3;
    //cudaSafeCall(hipMalloc((void**) &bf16_A, sizeof(__hip_bfloat16) * m * k * 3));
    //cudaSafeCall(hipMalloc((void**) &bf16_B, sizeof(__hip_bfloat16) * k * n * 3));

    convert_fp32_to_bf16(bf16_A, A, m*k);
    convert_fp32_to_bf16(bf16_B, B, k*n);

    float beta0 = *beta, beta1 = 1;
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, bf16_A, HIP_R_16BF, 
        lda, bf16_B, HIP_R_16BF, ldb, &beta0, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, bf16_A + m*k, HIP_R_16BF, 
        lda, bf16_B, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, bf16_A, HIP_R_16BF, 
        lda, bf16_B + k*n, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, bf16_A + m*k, HIP_R_16BF, 
        lda, bf16_B + k*n, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, bf16_A + m*k*2, HIP_R_16BF, 
        lda, bf16_B, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, alpha, bf16_A, HIP_R_16BF, 
        lda, bf16_B + k*n*2, HIP_R_16BF, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    
    //cudaSafeCall(hipFree(bf16_A));
    //cudaSafeCall(hipFree(bf16_B));
}
} // namespace ape
