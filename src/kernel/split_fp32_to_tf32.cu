#include "hip/hip_runtime.h"
#include "common.h"
#include "kernel.h"

namespace ape {
__global__ void kernel_split_fp32_to_tf32(float *dst, const float *src, size_t size) {
    uint32_t base = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    uint32_t step = 2 * blockDim.x * gridDim.x;
    for (uint32_t i = base; i < size; i += step) {
        float2 base = (float2 &)src[i];
        float2 buf[2];
        buf[0].x = base.x;
        buf[0].y = base.y;
        buf[1].x = (base.x - float(buf[0].x)) * 4096.0f;
        buf[1].y = (base.y - float(buf[0].y)) * 4096.0f;
        (float2 &)dst[i] = buf[0];
        (float2 &)dst[size + i] = buf[1];
    }
    return;
}

void split_fp32_to_tf32(float *dst, const float *src, size_t size) {
    dim3 grid_size(NUM_SM, 1);
    dim3 block_size(MAX_THREAD, 1);
    kernel_split_fp32_to_tf32<<<grid_size, block_size>>>(dst, src, size);
    cudaCheckError();
}
} // namespace ape