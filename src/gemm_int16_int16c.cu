#include "cuda_utils.h"
#include "kernel.h"

namespace ape {
// TODO: the interface of apeGemmINT16 seems contradictory with current hipblasGemmEx
void gemm_int16_int16c(ApeTrans transa, ApeTrans transb, int m, int n, int k, const int16_t *alpha, const int16_t *A, int lda,
                     const int16_t *B, int ldb, const int32_t *beta, int32_t *C, int ldc) {
    int8_t *buf, *buf_a, *buf_b;
    cudaSafeCall(hipMalloc((void **)&buf, sizeof(int8_t) * (m * k + k * n) * 2));
    buf_a = buf;
    buf_b = buf + m * k * 2;

    split_int16_to_int16c(buf_a, A, m * k);
    split_int16_to_int16c(buf_b, B, k * n);

    int alpha0 = *alpha * 256 * 256, alpha1 = *alpha * 256, alpha2 = *alpha;
    int beta0 = *beta, beta1 = 1;
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha0,
                                buf_a, HIP_R_8I, lda, buf_b, HIP_R_8I, ldb, &beta0, C, HIP_R_32I, ldc, HIPBLAS_COMPUTE_32I,
                                HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha1,
                                buf_a + m * k, HIP_R_8I, lda, buf_b, HIP_R_8I, ldb, &beta1, C, HIP_R_32I, ldc,
                                HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha1,
                                buf_a, HIP_R_8I, lda, buf_b + k * n, HIP_R_8I, ldb, &beta1, C, HIP_R_32I, ldc,
                                HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha2,
                                buf_a + m * k, HIP_R_8I, lda, buf_b + k * n, HIP_R_8I, ldb, &beta1, C, HIP_R_32I, ldc,
                                HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT));

    cudaSafeCall(hipFree(buf));
}
} // namespace ape