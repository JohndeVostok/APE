
#include "cuda_utils.h"
#include "kernel.h"

namespace ape {
void gemm_fp32_fp16(ApeTrans transa, ApeTrans transb, int m, int n, int k, const float *alpha, const float *A, int lda,
                      const float *B, int ldb, const float *beta, float *C, int ldc) {
    
    half *half_A = (half*)ape_buffer, *half_B = half_A + m * k * 2;
    //cudaSafeCall(hipMalloc((void**) &half_A, sizeof(half) * m * k * 2));
    //cudaSafeCall(hipMalloc((void**) &half_B, sizeof(half) * k * n * 2));

    split_fp32_to_fp16(half_A, A, m*k);
    split_fp32_to_fp16(half_B, B, k*n);

    float alpha0 = *alpha, alpha1 = *alpha / 4096.0f, beta0 = *beta, beta1 = 1;
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha0, half_A, HIP_R_16F, 
        lda, half_B, HIP_R_16F, ldb, &beta0, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha1, half_A + m*k, HIP_R_16F, 
        lda, half_B, HIP_R_16F, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    cublasSafeCall(hipblasGemmEx(ape_cublas_handle, hipblasOperation_t(transa), hipblasOperation_t(transb), m, n, k, &alpha1, half_A, HIP_R_16F, 
        lda, half_B + k*n, HIP_R_16F, ldb, &beta1, C, HIP_R_32F, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT));
    
    //cudaSafeCall(hipFree(half_A));
    //cudaSafeCall(hipFree(half_B));
}

} // namespace ape
